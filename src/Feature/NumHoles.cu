#include "hip/hip_runtime.h"
#include "NumHoles.h"
#include <iostream>

__global__ void kernel(State *states, double *returnVals)
{
    double returnVal = 0;
    auto sVector = states[threadIdx.x]->state;
    for(int z = 0; z < sVector[0].size(); z++)
    {
        for(int x = 0; x < sVector[0][0].size(); x++)
        {
            bool canBeHole = false;
            for(int y = 0; y < sVector.size(); y++)
            {
                if(!sVector[y][z][x])
                {
                    canBeHole = true;
                }
                else if (sVector[y][z][x] && canBeHole)
                {
                    returnVal++;
                    canBeHole = false;
                }
            }
            
        }
    }
    
    returnVals[theadIdx.x] = returnVal;
}

vector<double> NumHoles::calculateFeature(vector<State> s)
{
   //get size in bytes of needed memory space
   size_t stateSize = sizeof(State) * s.size();
   size_t returnSize = sizeof(double) * s.size();

   //setup pointers for host(CPU) and device(GPU)
   State *hostStates = &s[0];
   State *deviceStates;
   double *hostReturnVals = (double*)malloc(returnSize);
   double *deviceReturnVals;

   //allocate memory on the GPU
   hipMalloc(&deviceStates, stateSize);
   hipMalloc(&deviceReturnVals, returnSize);

   //copy memory from host to device
   hipMemcpy(deviceStates, hostStates, stateSize, hipMemcpyDeviceToHost);

   //run the GPU kernel
   kernel<<<1,s.size()>>>(deviceStates, deviceReturnVals);

   //copy memory back from device to host
   hipMemcpy(hostReturnVals, deviceReturnVals, returnSize, hipMemcpyHostToDevice);

   //create return vector from returned array
   vector<double> returnVec;
   for(int i = 0; i < s.size(); i++)
   {
       returnVec.push_back(hostReturnVals[i]);
   }

   //free up all memory from pointers
   free(hostStates);
   free(hostReturnVals);
   hipFree(deviceStates);
   hipFree(deviceReturnVals);

   return returnVec;
}