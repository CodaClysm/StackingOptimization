#include "hip/hip_runtime.h"
#include "MaxHeight.h"

__global__ void kernel(State *states, double *returnVals)
{
    double returnVal = 0;
    auto sVector = states[threadIdx.x]->state;
    for(int y = sVector.size()-1; y >= 0; y--)
    {
        for(int z = 0; z < sVector[y].size(); z++)
        {
            for(int x = 0; x < sVector[y][z].size(); x++)
            {
                if(sVector[y][z][x])
                {
                    returnVal += y;
                    break;
                }
            }
        }
    }
    returnVals[threadIdx.x] = returnVal;
}


vector<double> MaxHeight::calculateFeature(vector<State> s)
{
    //get size in bytes of needed memory space
    size_t stateSize = sizeof(State) * s.size();
    size_t returnSize = sizeof(double) * s.size();

    //setup pointers for host(CPU) and device(GPU)
    State *hostStates = &s[0];
    State *deviceStates;
    double *hostReturnVals = (double*)malloc(returnSize);
    double *deviceReturnVals;

    //allocate memory on the GPU
    hipMalloc(&deviceStates, stateSize);
    hipMalloc(&deviceReturnVals, returnSize);

    //copy memory from host to device
    hipMemcpy(deviceStates, hostStates, stateSize, hipMemcpyDeviceToHost);

    //run the GPU kernel
    kernel<<<1,s.size()>>>(deviceStates, deviceReturnVals);

    //copy memory back from device to host
    hipMemcpy(hostReturnVals, deviceReturnVals, returnSize, hipMemcpyHostToDevice);

    //create return vector from returned array
    vector<double> returnVec;
    for(int i = 0; i < s.size(); i++)
    {
        returnVec.push_back(hostReturnVals[i]);
    }

    //free up all memory from pointers
    free(hostStates);
    free(hostReturnVals);
    hipFree(deviceStates);
    hipFree(deviceReturnVals);

    return returnVec;
}