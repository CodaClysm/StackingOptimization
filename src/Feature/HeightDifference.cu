#include "hip/hip_runtime.h"
#include "HeightDifference.h"
#include "MaxHeight.h"
#include <iostream>

__global__ void kernel(State *states, double *returnVals)
{
    int maxHeight = 0;
    auto sVector = states[threadIdx.x].state;
    bool found = false;
    for(int y = sVector.size()-1; y >= 0; y--)
    {
        for(int z = 0; z < sVector[y].size(); z++)
        {
            for(int x = 0; x < sVector[y][z].size(); x++)
            {
                if(sVector[y][z][x])
                {
                    maxHeight = y;
                    found = true;
                    break;
                }
            }
            if(found)
            {
                break;
            }
        }
        if(found)
        {
            break;
        }
    }
    found = false;
    int minHeight = 0;
    for(int y = 0; y < sVector.size(); y++)
    {
        for(int z = 0; z < sVector[y].size(); z++)
        {
            for(int x = 0; x < sVector[y][z].size(); x++)
            {
                if(!sVector[y][z][x])
                {
                    minHeight = y;
                    found = true;
                    break;
                }
            }
            if(found)
            {
                break;
            }
        }
        if(found)
        {
            break;
        }
    }

    returnVals[threadIdx.x] = maxHeight - minHeight;
}


vector<double> HeightDifference::calculateFeature(vector<State> s)
{
    //get size in bytes of needed memory space
    size_t stateSize = sizeof(State) * s.size();
    size_t returnSize = sizeof(double) * s.size();

    //setup pointers for host(CPU) and device(GPU)
    State *hostStates = &s[0];
    State *deviceStates;
    double *hostReturnVals = (double*)malloc(returnSize);
    double *deviceReturnVals;

    //allocate memory on the GPU
    hipMalloc(&deviceStates, stateSize);
    hipMalloc(&deviceReturnVals, returnSize);

    //copy memory from host to device
    hipMemcpy(deviceStates, hostStates, stateSize, hipMemcpyDeviceToHost);

    //run the GPU kernel
    kernel<<<1,s.size()>>>(deviceStates, deviceReturnVals);

    //copy memory back from device to host
    hipMemcpy(hostReturnVals, deviceReturnVals, returnSize, hipMemcpyHostToDevice);

    //create return vector from returned array
    vector<double> returnVec;
    for(int i = 0; i < s.size(); i++)
    {
        returnVec.push_back(hostReturnVals[i]);
    }

    //free up all memory from pointers
    free(hostStates);
    free(hostReturnVals);
    hipFree(deviceStates);
    hipFree(deviceReturnVals);

    return returnVec;
}