#include "hip/hip_runtime.h"
#include "NumWells.h"
#include <iostream>
#include <vector>
#include <math.h>

__device__ void getColumnHeight(int z, int x, State s, int &height)
{
    auto sVector = s.state;
    if(z < 0 || z >= sVector[0].size() || x < 0 || x >= sVector[0][0].size())
    {
        height = -1;
        return;
    }
    for(int y = sVector.size()-1; y >= 0; y--)
    {
        if(sVector[y][z][x])
        {
            height = y;
            return;
        }
    }
    height = -1;
}

__global__ void kernel(State *states, double *returnVals)
{
    double returnVal = 0;
    auto sVector = states[threadIdx.x]->state;
    auto s = states[threadIdx.x];
    for(int z = 0; z < sVector[0].size(); z++)
    {
        for(int x = 0; x < sVector[0][0].size(); x++)
        {
            int h = 0;
            int currentColHeight = 0;
            getColumnHeight(z, x, s, currentColHeight);
            vector<int> heights;
            getColumnHeight(z+1,x+1,s, h)
            heights.push_back(h);
            getColumnHeight(z+1,x,s, h)
            heights.push_back(h);
            getColumnHeight(z+1,x-1,s,h)
            heights.push_back(h);
            getColumnHeight(z,x+1,s,h)
            heights.push_back(h);
            getColumnHeight(z,x-1,s,h)
            heights.push_back(h);
            getColumnHeight(z-1,x+1,s,h)
            heights.push_back(h);
            getColumnHeight(z-1,x,s,h)
            heights.push_back(h);
            getColumnHeight(z-1,x-1,s,h)
            heights.push_back(h);
            for(int i : heights)
            {
                if(i != -1 && abs(currentColHeight - i) > 2)
                {
                    returnVal ++;
                }
            }
        }
    }
    
    returnVals[theadIdx.x] = returnVal;
}

double NumWells::calculateFeature(State s)
{
    //get size in bytes of needed memory space
    size_t stateSize = sizeof(State) * s.size();
    size_t returnSize = sizeof(double) * s.size();

    //setup pointers for host(CPU) and device(GPU)
    State *hostStates = &s[0];
    State *deviceStates;
    double *hostReturnVals = (double*)malloc(returnSize);
    double *deviceReturnVals;

    //allocate memory on the GPU
    hipMalloc(&deviceStates, stateSize);
    hipMalloc(&deviceReturnVals, returnSize);

    //copy memory from host to device
    hipMemcpy(deviceStates, hostStates, stateSize, hipMemcpyDeviceToHost);

    //run the GPU kernel
    kernel<<<1,s.size()>>>(deviceStates, deviceReturnVals);

    //copy memory back from device to host
    hipMemcpy(hostReturnVals, deviceReturnVals, returnSize, hipMemcpyHostToDevice);

    //create return vector from returned array
    vector<double> returnVec;
    for(int i = 0; i < s.size(); i++)
    {
        returnVec.push_back(hostReturnVals[i]);
    }

    //free up all memory from pointers
    free(hostStates);
    free(hostReturnVals);
    hipFree(deviceStates);
    hipFree(deviceReturnVals);

    return returnVec;
}