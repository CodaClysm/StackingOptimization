#include "hip/hip_runtime.h"
#include "TransitionX.h"
#include <iostream>
#include <vector>
#include <math.h>

__global__ void kernel(State *states, double *returnVals)
{
    double returnVal = 0;
    auto sVector = states[threadIdx.x]->state;
    
    /*
    (if two horizontally adjacent cells are not both occupied or free, they count as a transition cell)
    Here, a transition occurs for each edge of an occupied cell that is adjacent to an unoccupied cell along the X axis. The walls count as occupied cells. 
    */
    for(int y = 0; y<sVector.size(); y++)
    {
        for(int z = 0; z < sVector[0].size(); z++)
        {
            for(int x = 0; x < sVector[0][0].size(); x++)
            {
                if(!sVector[y][z][x])
                {
                    if(x-1 < 0)
                    {
                        returnVal++;
                        if(sVector[y][z][x+1])
                        {
                            returnVal++;
                        }
                    }
                    else if(x+1 >= sVector[0][0].size())
                    {
                        returnVal++;
                        if(sVector[y][z][x-1])
                        {
                            returnVal++;
                        }
                    }
                    else
                    {
                        if(sVector[y][z][x-1])
                        {
                            returnVal++;
                        }
                        if(sVector[y][z][x+1])
                        {
                            returnVal++;
                        }
                    }
                }              
            }
        }
    }

    returnVals[theadIdx.x] = returnVal;
}

vector<double> TransitionX::calculateFeature(vector<State> s)
{
    //get size in bytes of needed memory space
    size_t stateSize = sizeof(State) * s.size();
    size_t returnSize = sizeof(double) * s.size();

    //setup pointers for host(CPU) and device(GPU)
    State *hostStates = &s[0];
    State *deviceStates;
    double *hostReturnVals = (double*)malloc(returnSize);
    double *deviceReturnVals;

    //allocate memory on the GPU
    hipMalloc(&deviceStates, stateSize);
    hipMalloc(&deviceReturnVals, returnSize);

    //copy memory from host to device
    hipMemcpy(deviceStates, hostStates, stateSize, hipMemcpyDeviceToHost);

    //run the GPU kernel
    kernel<<<1,s.size()>>>(deviceStates, deviceReturnVals);

    //copy memory back from device to host
    hipMemcpy(hostReturnVals, deviceReturnVals, returnSize, hipMemcpyHostToDevice);

    //create return vector from returned array
    vector<double> returnVec;
    for(int i = 0; i < s.size(); i++)
    {
        returnVec.push_back(hostReturnVals[i]);
    }

    //free up all memory from pointers
    free(hostStates);
    free(hostReturnVals);
    hipFree(deviceStates);
    hipFree(deviceReturnVals);

    return returnVec;
}